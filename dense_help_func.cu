#ifndef DENSE_HELP_FUNC
#define DENSE_HELP_FUNC

#include <hipblas.h>
#define checkCudaErrors(func)				\
{									\
    hipError_t e = (func);			\
    if(e != hipSuccess)						                \
        printf ("%s %d CUDA: %s\n", __FILE__,  __LINE__, hipGetErrorString(e));		\
}

static const char *_cuBlasGetErrorEnum(hipblasStatus_t error)
{
    switch (error)
    {
        case HIPBLAS_STATUS_SUCCESS:
            return "CUBLAS_STATUS_SUCCESS";

        case HIPBLAS_STATUS_NOT_INITIALIZED:
            return "CUBLAS_STATUS_NOT_INITIALIZED";

        case HIPBLAS_STATUS_ALLOC_FAILED:
            return "CUBLAS_STATUS_ALLOC_FAILED";

        case HIPBLAS_STATUS_INVALID_VALUE:
            return "CUBLAS_STATUS_INVALID_VALUE";

        case HIPBLAS_STATUS_ARCH_MISMATCH:
            return "CUBLAS_STATUS_ARCH_MISMATCH";

        case HIPBLAS_STATUS_MAPPING_ERROR:
            return "CUBLAS_STATUS_MAPPING_ERROR";

        case HIPBLAS_STATUS_EXECUTION_FAILED:
            return "CUBLAS_STATUS_EXECUTION_FAILED";

        case HIPBLAS_STATUS_INTERNAL_ERROR:
            return "CUBLAS_STATUS_INTERNAL_ERROR";
    }
    return "<unknown>";
}


#define checkCuBlasErrors(func)				\
{									\
    hipblasStatus_t e = (func);			\
    if(e != HIPBLAS_STATUS_SUCCESS)						                \
        printf ("%s %d CuBlas: %s", __FILE__,  __LINE__, _cuBlasGetErrorEnum(e));		\
}



#endif

// void errorHandler(cudaError_t error, const char *file, const int line)
// {
//     printf("CUDA error %d at %s:%d\n", error, file, line);
//     exit(EXIT_FAILURE);
// }

// #define CHECK_CUDNN_ERROR(err) \
//     if (err != CUDNN_STATUS_SUCCESS) { \
//         fprintf(stderr, "CUDNN error: %s\n", cudnnGetErrorString(err)); \
//         exit(EXIT_FAILURE); \
//     }
